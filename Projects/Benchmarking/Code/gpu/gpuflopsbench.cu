#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

/*
 
Aim         : To benchmark the GPU in terms of FLOPS and IOPS with.

Description : This program finds the FLOPS and IOPS of GPU. 
              The gpuIntFun and gpuFloatFun are the functions which perform the operation on GPU environment. 
              The host variables are first declared and initialized. Same number of variables are allocated memory on GPU, 
              and data is copied fro host to device variables.
              The result variable is the copied back to host.

Contributor : Vivek Pabani (A20332117)
*/


#define LIMIT 500

__global__ void gpuIntFun(int *a, int *b, int *c)
{
    int tid = blockIdx.x*blockDim.x+threadIdx.x;
    c[tid] = a[tid] + b[tid];
}
__global__ void gpuFloatFun(float *a, float *b, float *c)
{
    int tid = blockIdx.x*blockDim.x+threadIdx.x;
    c[tid] = a[tid] + b[tid];
}

int main(void)
{
    int numberOfThreads;
    int blocks, threads;
    int i=0;
    int choice=0;
    
    double time_s;
    long start_time,end_time;
    struct timeval start,stop;

    struct hipDeviceProp_t gpuDetails;
    int numberOfDevices, device=0;

    hipError_t cudaResultCode = hipGetDeviceCount(&numberOfDevices);

//To get device info on run time

    for (device = 0; device < numberOfDevices; ++device) {
        hipGetDeviceProperties(&gpuDetails, device);

//To set device variables to be used for thread creation.

        if (gpuDetails.major != 9999)
        {    
            blocks=gpuDetails.multiProcessorCount;
            threads=gpuDetails.maxThreadsPerMultiProcessor;
            numberOfThreads=blocks * threads;
        }
    }

    printf("Processor Count %d\n",blocks);
    printf("Thread per Processor %d\n",threads);
    printf("Total Threads %d\n",numberOfThreads);

    /*
    choice 
    0 - IOPS
    1 - FLOPS
    */
    
    for(choice=0; choice<2; ++choice)
    {
        if(choice == 0)
        {
            int *host_a, *host_b, *host_c;
            int *dev_a, *dev_b, *dev_c;

            //assign memory to host variables.
            
            host_a = (int*) malloc(numberOfThreads*sizeof(int)); 
            host_b = (int*) malloc(numberOfThreads*sizeof(int)); 
            host_c = (int*) malloc(numberOfThreads*sizeof(int)); 

            //Initialize host variables.
            
            for(i=0;i<numberOfThreads;++i)
            {
                host_a[i] = (i*25)+25;
                host_b[i] = (i*36)+36; 
            }

            //assign memory to device variables

            hipMalloc((void**)&dev_a, numberOfThreads*sizeof(int));
            hipMalloc((void**)&dev_b, numberOfThreads*sizeof(int));
            hipMalloc((void**)&dev_c, numberOfThreads*sizeof(int));

            // copy variables to device memory
            
            hipMemcpy(dev_a,host_a,numberOfThreads*sizeof(int),hipMemcpyHostToDevice);
            hipMemcpy(dev_b,host_b,numberOfThreads*sizeof(int),hipMemcpyHostToDevice);

            //Calculation Time Starts
            
            gettimeofday(&start,NULL);
    
            start_time=start.tv_sec*1000000 + start.tv_usec;

            for (i=0; i<LIMIT; ++i)
            {
                gpuIntFun<<<blocks,threads>>>(dev_a,dev_b,dev_c);

            }
    
            gettimeofday(&stop,NULL);

            //Calculation Time Ends

            end_time=stop.tv_sec*1000000 + stop.tv_usec;//get end time

            hipMemcpy(host_c,dev_c,numberOfThreads*sizeof(int),hipMemcpyDeviceToHost);

            time_s=end_time-start_time;
            
            printf("\nTime taken: %lf",time_s);
            printf("\nIOPS: %lf",(double)(LIMIT*numberOfThreads*1.024*0.1048576)/(time_s*2.0));

            hipFree(dev_a);
            hipFree(dev_b);
            hipFree(dev_c);
        }
       
        else if(choice == 1)
        {
            float *host_a, *host_b, *host_c;
            float *dev_a, *dev_b, *dev_c;

            host_a = (float*) malloc(numberOfThreads*sizeof(float)); 
            host_b = (float*) malloc(numberOfThreads*sizeof(float)); 
            host_c = (float*) malloc(numberOfThreads*sizeof(float)); 

            for(i=0;i<numberOfThreads;++i)
            {
                host_a[i] = (i*25.5)+25.5;
                host_b[i] = (i*36.6)+36.6; 
            }

            hipMalloc((void**)&dev_a, numberOfThreads*sizeof(float));
            hipMalloc((void**)&dev_b, numberOfThreads*sizeof(float));
            hipMalloc((void**)&dev_c, numberOfThreads*sizeof(float));

            hipMemcpy(dev_a,host_a,numberOfThreads*sizeof(float),hipMemcpyHostToDevice);
            hipMemcpy(dev_b,host_b,numberOfThreads*sizeof(float),hipMemcpyHostToDevice);

            gettimeofday(&start,NULL);
    
            start_time=start.tv_sec*1000000 + start.tv_usec;

            for (i=0; i<LIMIT; ++i)
            {
                gpuFloatFun<<<blocks,threads>>>(dev_a,dev_b,dev_c);

            }
    
            gettimeofday(&stop,NULL);
            end_time=stop.tv_sec*1000000 + stop.tv_usec;//get end time

            hipMemcpy(host_c,dev_c,numberOfThreads*sizeof(int),hipMemcpyDeviceToHost);

            time_s=end_time-start_time;
            
            printf("\nTime taken: %lf",time_s);
            printf("\nGFLOPS: %lf\n",(double)(LIMIT*numberOfThreads*1.024*0.1048576)/(time_s*2.0));

            hipFree(dev_a);
            hipFree(dev_b);
            hipFree(dev_c);

        }    


    }
    return 0;

}