#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

/*
 
Aim         : To benchmark the GPU in terms of Read and Write Bandwidth with different types of block sizes.

Description : This program finds Read and Write Memory Bandwidth of GPU.
              The main function creates memory on CPU and copies to GPU, and then read memory from GPU into CPU.
              This operation is done for three types of bloxk sizes : 1B, 1KB, 1MB.
                  
Contributor : Vivek Pabani (A20332117)
*/


int main(void)
{
    double time_s;
    long start_time,end_time;
    struct timeval start,stop;

    char *host_memory;//, *dev_memory;
    
    long blockSize[3] = {1,1024,1048576};
    long limit[3] = {1000000,100000,100000};
    char blockTypes[3][20] = {"B","KB","MB"};
    char operationTypes[2][20] = {"Write","Read"};
    int i=0, j=0, k=0, numberOfBlocks = 3, numberOfOperations = 2;
            
    /*
    Operations :
    0:Write, 
    1 Read, 
    */

    for(j=0; j<numberOfOperations; ++j)
    {
        printf("\n----Operation = %s---- \n\n",operationTypes[j]);
        printf("Block Size\tTotal Data\tTotal Time ms\tBandwidth MBPS\n\n");
  
        /*
         Block Sizes :
         0 : 1 B 
         1 : 1 KB
         2 : 1 MB
         
         */
               
        for(k=0; k<numberOfBlocks; ++k)
        {
            printf("1 %s\t\t%7d %2s\t",blockTypes[k],limit[k],blockTypes[k]);

            char *dev_memory;
                    
            /* Assign the Host Memory*/    
            
            host_memory = (char*) malloc(blockSize[k]*sizeof(char)); 

            /* Assign the Device Memory*/    

            hipMalloc((void**)&dev_memory, blockSize[k]*sizeof(char));

            /* Write Operation*/    
            
            if(j==0)
            {
                memset(host_memory,'a',blockSize[k]);

                /* Time calculation for main operation starts */    

                gettimeofday(&start,NULL);
    
                for (i=0; i<limit[k]; ++i)
                {
                    hipMemcpy(dev_memory,&host_memory[i],blockSize[k]*sizeof(char),hipMemcpyHostToDevice);
                }
    
                gettimeofday(&stop,NULL);

                /* Time calculation for main operation ends */    

            }
            
            /* Read Operation*/    

            else if (j==1)
            {
                hipMemset(dev_memory,'a',blockSize[k]);

                /* Time calculation for main operation starts */    
    
                gettimeofday(&start,NULL);
    
                for (i=0; i<limit[k]; ++i)
                {
                    hipMemcpy(host_memory,&dev_memory[i],blockSize[k]*sizeof(char),hipMemcpyDeviceToHost);
                }
    
                gettimeofday(&stop,NULL);
                
                /* Time calculation for main operation starts */    

            }
        
            start_time=start.tv_sec*1000000 + start.tv_usec;
            end_time=stop.tv_sec*1000000 + stop.tv_usec;//get end time

            time_s=end_time-start_time;
            printf("%8.3f\t%9.5f\n",(time_s/1000),(blockSize[k]*limit[k]*1000)/(time_s*1024*1024));

            hipFree(dev_memory);
        }
    }
        
    return 0;
}